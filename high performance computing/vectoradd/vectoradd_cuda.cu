//nsys nvprof ./vectoradd_cuda


#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 1000000000

void  initVector(float *u, int n, float c) {
  int i;
  for (i=0; i<n; i++)
      u[i] = c;
}

__global__ void gpuVectAdd(float *u, float *v, float *z) 
{
  // define index
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  // check that the thread is not out of the vector boundary
  if (i >= N ) return;

  int index = i; 
  // write the operation for the sum of vectors 
  z[index] = u[index] + v[index];
}


int main(int argc, char *argv[]) {

  // size of vectors
  //const int N = 1000;

  hipEvent_t start, stop, start1, stop1;     // using cuda events to measure time
  float elapsed_time_ms;       // which is applicable for asynchronous code also

  // allocate memory on host
  float * u = (float *) malloc(N * sizeof(float));
  float * v = (float *) malloc(N * sizeof(float));
  float * z = (float *) malloc(N * sizeof(float));

  initVector((float *) u, N, 1.0);
  initVector((float *) v, N, 2.0);
  initVector((float *) z, N, 0.0);

  hipEventCreate( &start );   hipEventCreate( &start1 );  // instrument code to measure start time
  hipEventCreate( &stop ); hipEventCreate( &stop1 );

  hipEventRecord( start1, 0 );
    // allocate memory on device
  float *u_dev, *v_dev, *z_dev;
  hipMalloc((void **) &u_dev, N*sizeof(float));
  hipMalloc((void **) &v_dev, N*sizeof(float));
  hipMalloc((void **) &z_dev, N*sizeof(float));

  // copy data from host to device
  hipMemcpy(u_dev, u, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(v_dev, v, N*sizeof(float), hipMemcpyHostToDevice);

  dim3 block(512);
  dim3 grid((N-1)/block.x + 1);

  hipEventRecord( start, 0 );

  // define the execution configuration
  gpuVectAdd<<<grid, block>>>(u_dev, v_dev, z_dev);

  hipEventRecord( stop, 0 );     // instrument code to measue end time
  hipEventSynchronize( stop );

  // copy data from device to host
  hipMemcpy(z, z_dev, N*sizeof(float), hipMemcpyDeviceToHost);

  hipEventRecord( stop1, 0 );     // instrument code to measue end time
  hipEventSynchronize( stop1 );

  hipEventElapsedTime( &elapsed_time_ms, start, stop );

  printf("%f %f %f\n", z[0], z[1], z[1]);
  printf("Time to calculate results: %f ms.\n", elapsed_time_ms);  // print out execution time
  hipEventElapsedTime( &elapsed_time_ms, start1, stop1 );
  printf("Time with I/O: %f ms.\n", elapsed_time_ms);


  // free resources on device
  hipFree(u_dev);
  hipFree(v_dev);
  hipFree(z_dev);

  // free resources on host
  free(u);
  free(v);
  free(z);

  return 0;
}

